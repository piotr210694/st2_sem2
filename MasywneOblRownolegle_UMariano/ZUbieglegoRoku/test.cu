/* 

ZWYKLE:
gcc test.c -o test
./test

CUDA:
test.cu
nvcc test.cu -o test
./test

*/

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

__global__
void funkcja_testowa(){

}

int main(){
	funkcja_testowa<<<1,1>>>();
	printf("CUDA error: %s\n", hipGetErrorString(hipGetLastError()));
}
